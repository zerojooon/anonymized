#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <vector>
#include <algorithm>
#include <chrono>

#include "mpi.h"
#include "param.h"
#include "util.h"
#include "lsh.h"

using namespace std;

int main(int argc, char* argv[]) {
    MPI_Init(&argc, &argv); // Initialize MPI
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank); // Get rank of the process
    MPI_Comm_size(MPI_COMM_WORLD, &size); // Get total number of processes

    int deviceCount = 0;
    hipError_t cuda_status = hipGetDeviceCount(&deviceCount); // Get number of GPUs
    if (cuda_status != hipSuccess) {return 1;}
    hipSetDevice(rank % deviceCount); // Assign GPU to the process

    if (argc != 3) {
        if (rank == 0)
            std::cerr << "Directory path require" << std::endl;
        MPI_Finalize();
        return 1;
    }
    std::string folderPath = argv[1]; // Input folder path
    std::string outputPath = argv[2]; // Output folder path

    vector<string> file_list;
    getFileList(file_list, folderPath); // Get list of files in the input folder
    sort(file_list.begin(), file_list.end()); // Sort files alphabetically

    int num_file = file_list.size(); // Total number of files
    int files_per_process = num_file / size;    
    int extra = num_file % size;               

    int start_index = rank * files_per_process + std::min(rank, extra);  // Start index for the process
    int end_index = start_index + files_per_process ; // End index for the process

    if (rank < extra) end_index++;      
    // Files assigned per process
    files_per_process = end_index - start_index ;

    // Initialize parameters for LSH
    int num_hash = NUM_HASH;
    int b = BUCKET;
    int shingle_len = SHINGLE_LEN;
    init_lsh_cuda(num_hash, shingle_len, b, 777984, 0.8, num_file); //num_hash, shingle_len, random_seed
    if(!rank) generate_file_init(outputPath);

    int *file_size= (int*)malloc(sizeof(int) * num_file);

    // Generate Minhash signature matrix and Calculate the bucket IDs of each band.
    if (rank == 0) {
        std::cout << "Start Minhash Generation.." << std::endl;
    }

    auto time1 = std::chrono::high_resolution_clock::now();

    for (int i=start_index; i < end_index; i++) {
        const string &fp=file_list[i];
        lsh_cuda(fp, outputPath, file_size[i], i, num_file);
    }

    // Gather file sizes from all processes    
    AllgatherFileSize(size, files_per_process, file_size);

    auto time2 = std::chrono::high_resolution_clock::now();
    
    std::chrono::duration<double> elapsed1 = time2 - time1;
    
    // Calculate time taken for MinHash
    if (rank == 0) {
        std::cout << "Min Hash total time: " << elapsed1.count() << " seconds" << std::endl;
        std::cout << "  - File read time: " << get_total_file_read_time_lsh() << " seconds" << std::endl;
        std::cout << "  - Computation time(c2g): " << c2g() << " seconds" << std::endl;
        std::cout << "  - Computation time: " << get_total_computation_time_lsh() << " seconds" << std::endl;
        std::cout << "  - Computation time(g2c): " << g2c() << " seconds" << std::endl;
        std::cout << "  - File write time: " << get_total_file_write_time_lsh() << " seconds\n" << std::endl;
    }

    // Comparison phase
    if (rank == 0) {
        std::cout << "Start Comparison.." << std::endl;
    }

    // (when file offloading is disabled) 
    // Gathers the hash results from all processes into the total_hash_result array
    AllgatherHashResult(rank, size, files_per_process, start_index);

    time1 = std::chrono::high_resolution_clock::now();
    compare_lsh_cuda(file_list, outputPath, num_file, file_size, rank, size);
    MPI_Barrier(MPI_COMM_WORLD);
    time2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = time2 - time1;
    if (rank == 0) {
        std::cout << "\nComparison total time: " << elapsed2.count() << " seconds" << std::endl;
        print_cmp_time_lsh();
    }

    time1 = std::chrono::high_resolution_clock::now();
    merge_union(rank, size); // Merge results across processes
    time2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = time2 - time1;

    MPI_Barrier(MPI_COMM_WORLD);

    if (rank == 0) {
        std::cout << "Saving the final cleaned dataset.." << std::endl;
    }
    delete_hash_result(outputPath); // remove the binary hash result files
    time1 = std::chrono::high_resolution_clock::now();
    for (int i=start_index; i < end_index; i++) {
        const string &fp=file_list[i];
        generate_file(fp, i, outputPath);  // save the final deduplicated dataset 
    }
    MPI_Barrier(MPI_COMM_WORLD);
    time2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed4 = time2 - time1;
    
    // Print total times for all phases
    if (rank == 0) {
        printf("==================================================\n");
        std::cout << "Min Hash total time: " << elapsed1.count() << " seconds" << std::endl;
        std::cout << "Comparison total time: " << elapsed2.count() << " seconds" << std::endl;
        std::cout << "Union total time: " << elapsed3.count() << " seconds" << std::endl;
        std::cout << "File generate time: " << elapsed4.count() << " seconds" << std::endl;
        std::cout << "Total time: " << elapsed1.count()+elapsed2.count()+elapsed3.count()+elapsed4.count() << " seconds" << std::endl;
    }
    finalize_lsh();
    MPI_Finalize();
    return 0;
}
